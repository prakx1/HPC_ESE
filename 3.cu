
#include<stdio.h>
#include<hip/hip_runtime.h>
#define row1 10 
#define col1 10  

#define col2 10 

__global__ void matrix_multiply(int *l,int *m, int *n)
{
    int x=blockIdx.x;
    int y=blockIdx.y;
    int k;
  
n[col2*y+x]=0;
for(k=0;k<col1;k++)
   {
    n[col2*y+x]=n[col2*y+x]+l[col1*y+k]*m[col2*k+x];
   }
}

int main()
{
    int a[row1][col1];
    int b[col2];
    int c[row1][col2];
    int *d,*e,*f;
    int i,j;

    for(i=0;i<row1;i++)
    {
        for(j=0;j<col1;j++)
            {
                a[i][j]=i;
            }
    }
    
        for(i=0;i<col2;i++)
        {
           b[i]=2;
        }
    hipMalloc((void **)&d,row1*col1*sizeof(int));
    hipMalloc((void **)&e,col2*sizeof(int));
    hipMalloc((void **)&f,row1*col2*sizeof(int));

 hipMemcpy(d,a,row1*col1*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(e,b,col2*sizeof(int),hipMemcpyHostToDevice);

dim3 grid(col2,row1);
    matrix_multiply<<<grid,1>>>(d,e,f);

 hipMemcpy(c,f,row1*col2*sizeof(int),hipMemcpyDeviceToHost);

 printf("\nVector:\n");
 for(i=0;i<col2;i++)
 printf("%d\t",b[i]);

 printf("\nMatrix:\n");
 for(i=0;i<row1;i++)
 {
     for(j=0;j<col1;j++)
     {
         printf("%d\t",a[i][j]);
     }
  printf("\n");
 }
  printf("\nProduct of vector and matrix:\n ");
 int sum=0;
    for(i=0;i<row1;i++)
    {   sum=0;
        for(j=0;j<col2;j++)
        {     sum+=c[i][j];
             
        }
      printf("%d\t",sum);
        printf("\n");
    }
    hipFree(d);
    hipFree(e);
    hipFree(f);
 
  

    return 0;

}



